#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


#include <string.h>

#define MATRIX_SIZE 2056
#define BLOCK_SIZE 32
#define CLOCKS_PER_SEC 1000000

__global__ void multiply_matrices(int *matrix_a, int *matrix_b, int *matrix_result) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < MATRIX_SIZE && col < MATRIX_SIZE) {
        int sum = 0;
        for (int i = 0; i < MATRIX_SIZE; i++) {
            sum += matrix_a[row * MATRIX_SIZE + i] * matrix_b[i * MATRIX_SIZE + col];
        }
        matrix_result[row * MATRIX_SIZE + col] = sum;
    }
}

void print_matrix(int *matrix) {
    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; i++) {
        printf("%d ", matrix[i]);
        if ((i + 1) % MATRIX_SIZE == 0) {
            printf("\n");
        }
    }
    printf("\n");
}

int main() {
    int *host_matrix_a, *host_matrix_b, *host_matrix_result;
    int *device_matrix_a, *device_matrix_b, *device_matrix_result;
    size_t size = MATRIX_SIZE * MATRIX_SIZE * sizeof(int);
    srand(time(NULL));

    clock_t start = clock();

    host_matrix_a = (int *)malloc(size);
    host_matrix_b = (int *)malloc(size);
    host_matrix_result = (int *)malloc(size);

    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; i++) {
        host_matrix_a[i] = rand() % 10;
        host_matrix_b[i] = rand() % 10;
    }

    printf("Matrix A:\n");
    print_matrix(host_matrix_a);
    printf("Matrix B:\n");
    print_matrix(host_matrix_b);

    hipMalloc(&device_matrix_a, size);
    hipMalloc(&device_matrix_b, size);
    hipMalloc(&device_matrix_result, size);

    hipMemcpy(device_matrix_a, host_matrix_a, size, hipMemcpyHostToDevice);
    hipMemcpy(device_matrix_b, host_matrix_b, size, hipMemcpyHostToDevice);

    memset(host_matrix_result, 0, size);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(MATRIX_SIZE / threadsPerBlock.x, MATRIX_SIZE / threadsPerBlock.y);

    multiply_matrices<<<numBlocks, threadsPerBlock>>>(device_matrix_a, device_matrix_b, device_matrix_result);

    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
    }

    hipMemcpy(host_matrix_result, device_matrix_result, size, hipMemcpyDeviceToHost);

    printf("Matrix Result:\n");
    print_matrix(host_matrix_result);

    free(host_matrix_a);
    free(host_matrix_b);
    free(host_matrix_result);
    hipFree(device_matrix_a);
    hipFree(device_matrix_b);
    hipFree(device_matrix_result);

    printf("\n-------------------\n");
    printf("Execution Time (CPU): %f seconds\n", ((double) clock() - start) / CLOCKS_PER_SEC);

    return 0;
}
